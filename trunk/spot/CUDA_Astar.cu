#include "hip/hip_runtime.h"
﻿#include "cudaheader.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <cstdlib>
#define PSIZE 32 
#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)
inline void __checkCudaErrors(hipError_t err, const char *file, const short line )
{
    if(hipSuccess != err)
    {
        fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",file, line, (short)err, hipGetErrorString( err ) );
        exit(-1);        
    }
}
__device__ void push(CONFIGSTACK *set, int *tail, CONFIGSTACK *node)
{
	int pos;
	pos=atomicCAS(tail, SETSIZE-1, 0);
	if (*tail==pos)
		pos=atomicAdd(tail,1);
	set[pos]=*node;
}
__device__ CONFIGSTACK  pop (CONFIGSTACK *set, int *head, int *tail)
{
	int pos;
	if (*head!=*tail)
	{
		pos=atomicCAS(head, SETSIZE-1, 0);
		if (*head==pos)
			pos=atomicAdd(head, 1);
		return set[pos];//always pop the 0 + offset element
	}
}
__device__ bool find(CONFIGSTACK *set, CONFIGSTACK *state, int *head, int *tail)
{
	for (int i=*head;i!=*tail;)
	{
		int j;
		bool flag=true;
		for (j=0;j<VSIZE;++j)
		{
			if (set[i].configurations[j]!=state->configurations[j])
			{
				flag=false;
				break;
			}
		}
		if (flag)
			return true;
		if (i!=SETSIZE-1)
			i++;
		else
			i=0;
	}
	return false;
}
__global__ void evaluate (//Lock lock_o, Lock lock_c, Lock lock_s
						CONFIGSTACK *buffer, 
						DAG_CUDA dag, CONFIGSTACK *openSet, int *osHead, int *osTail, CONFIGSTACK *closeSet, int *csHead, int* csTail, CONFIGSTACK *solutionSet, int *solutionSize, 
						int flag, int types, float *probestTime, float OnDemandLag, 
						float deadline, float meet_dl, int randomsize, float *priceOnDemand, float globalBestCost, int *gv, int *inEdgeCount, int *in_edge_own)
{
	int bid=blockIdx.x;
	int tid=threadIdx.x;
	//int *inEdgeCount, *in_edge_own;
	//inEdgeCount=(int*)malloc(sizeof(int)*VSIZE);
	//in_edge_own=(int*)malloc(sizeof(int)*ESIZE*VSIZE);

	//__shared__ int inEdgeCount[VSIZE];
	//__shared__ int in_edge_own[ESIZE][VSIZE];//worst case
	__shared__ int totalcost;
	__shared__ int count;
	__shared__ CONFIGSTACK currentNode;
	__shared__ CONFIGSTACK  child;
	
	if (tid==0)
	{
		currentNode=buffer[bid];
		count=0;
	}	//fetch the node from buffer
	/*
	if (tid<VSIZE)
	{
            inEdgeCount[tid]=0;
            for (int i=1;i<2*ESIZE;i+=2)
            {
                    if (dag.edge[i]==tid)
                    {
                            in_edge_own[inEdgeCount[tid]+ESIZE*tid]=dag.edge[i-1];
                            inEdgeCount[tid]++;
                    }
            }
    	}
	__syncthreads();
	*/
	int len=0;
	int lc=0;
	float cumulativeTime[VSIZE];//accuracy
	while (len+tid<randomsize)
	{
		for (int i=0;i<VSIZE;++i)
		{
			cumulativeTime[i]=0;
		}
		for (int i=0;i<VSIZE;++i)
		{
			int config=currentNode.configurations[i];
			float tmpTime=0;
			if (inEdgeCount[i]==0)
			{
				cumulativeTime[i]=probestTime[i*randomsize*TYPES+config*randomsize+len+tid] + OnDemandLag;
			}
			else
			{
				for (int j=0;j<inEdgeCount[i];++j)
				{
					tmpTime= tmpTime>cumulativeTime[in_edge_own[j+ESIZE*i]]?tmpTime:cumulativeTime[in_edge_own[j+ESIZE*i]];
				}
				cumulativeTime[i]=tmpTime+probestTime[i*randomsize*TYPES + currentNode.configurations[i] * randomsize + len + tid] + OnDemandLag;
			}
		}		
		
		if(cumulativeTime[VSIZE-1]<=deadline)
		{
			lc++;
		}
		
		len+=blockDim.x;
	}
	atomicAdd(&count, lc);
	__syncthreads();
	if (tid==0)
	{
		if ((float)count / (float)randomsize >= meet_dl)
		{
		int fv=(int)(currentNode.fvalue*1000.0);
		if (fv<=atomicMin(gv, fv))
		{
			push(solutionSet, solutionSize, &currentNode);
		}
		}
	}
	if (tid==32)
	{
		push(closeSet, csTail, &currentNode);
		child=currentNode;
		child.taskno=currentNode.taskno + 1;
	}
	
	__syncthreads();	
	
	if (child.taskno < VSIZE)
	{
		for (int t=currentNode.configurations[child.taskno]+1;t<types;t++)
		{
			if (tid ==0)
			{
				child.configurations[child.taskno]=t;
			}
			__syncthreads();
			if (flag == deadlineapp)
			{
					if (tid==0)
						totalcost=0;
					for (int i=0;i<VSIZE;++i)
					{
						float taskcost=0;
						int len=0;
						while(len+tid<randomsize)//for (int j=0;j<randomsize;++j)
						{
							taskcost+=ceilf((probestTime[i*randomsize*TYPES+child.configurations[i]*randomsize+len+tid]+OnDemandLag)/3600.0)*priceOnDemand[child.configurations[i]];//edit
							len+=blockDim.x;
						}
						taskcost*=1000.0;
						atomicAdd(&totalcost, (int)taskcost);
					}
					__syncthreads();
					if (tid==0)
					{
						totalcost = (int)(1.0 * totalcost / randomsize);
						if (!( totalcost >= *gv || find(closeSet, &child, csHead, csTail))){
							child.fvalue=totalcost*1.0/1000;
							push(openSet, osTail, &child);
						}
					}
			}
		}
	}

}
__device__ bool equal(CONFIGSTACK a, CONFIGSTACK b)
{
	if (a.fvalue!=b.fvalue)
		return false;
	for (int i=0;i<VSIZE;++i)
		if (a.configurations[i]!=b.configurations[i])
			return false;
	return true;
}
#define buffer_size_max 16
__global__ void top_k(int *buffer_size, CONFIGSTACK *buffer, CONFIGSTACK *openSet, int *osHead, int *osTail)
{
	//to edit
	int bid=blockIdx.x;
	int tid=threadIdx.x;
	int sizeOpenSet=*osTail>=*osHead?(*osTail-*osHead):(SETSIZE-*osHead+*osTail);
	extern __shared__ float pos[];
	if (sizeOpenSet<buffer_size_max)
	{
		//if (tid < buffer_size_max)
		if (bid==0 && tid==0)
		{
			for (int i=0;i<sizeOpenSet;++i)
				buffer[i]=pop(openSet, osHead, osTail);
			*buffer_size=sizeOpenSet;
		}
	}
	else //if (sizeOpenSet<2*buffer_size_max)
	{
		pos[tid*2]=*osHead+bid*sizeOpenSet/gridDim.x+tid;
		pos[tid*2+1]=openSet[(int)pos[tid*2]].fvalue;
		int len=blockDim.x;
		while (len+pos[tid*2]<*osHead+(bid+1)*sizeOpenSet/gridDim.x)
		{
			if (openSet[len+(int)pos[tid*2]].fvalue<pos[tid*2+1])
			{
				pos[tid*2+1]=openSet[len+(int)pos[tid*2]].fvalue;
				pos[tid*2]+=len;
			}
			len+=blockDim.x;
		}
		len=blockDim.x/2;
		__syncthreads();
		while (len>=1)
		{
			if (tid<len)
			{
				if (pos[(tid+len)*2+1]<pos[tid*2+1])
				{
					pos[tid*2+1]=pos[(tid+len)*2+1];
					pos[tid*2]=pos[(tid+len)*2];
				}
			}
			len/=2;
			__syncthreads();
		}
		if (tid==0)
		{	
		buffer[bid]=openSet[(int)pos[0]];
		CONFIGSTACK temp;
		temp=pop(openSet, osHead, osTail);
		if (!equal(temp, buffer[bid]))
			openSet[(int)pos[0]]=temp;
		*buffer_size=buffer_size_max;
		}
	}
}
extern "C" void Astar_CUDA (int randomsize, int *vertex, int *edge, int v_size, int e_size,int flag,int types, 
				 float *probestTime[VSIZE],float OnDemandLag, const float *priceOnDemand, float globalBestCost, float deadline,float meet_dl,CONFIGSTACK initstate)
{
	checkCudaErrors(hipSetDevice(0));
	//Lock lock_o, lock_c, lock_s, 
	printf ("CUDA function started\n");
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);	
	hipEventRecord(start, 0);
	CONFIGSTACK *openSet, *closeSet;
	int *osTail, *osHead, *csTail, *csHead, *d_osTail, *d_osHead;

	checkCudaErrors(hipHostAlloc((void**)&osTail,sizeof(int),hipHostMallocMapped));
	checkCudaErrors(hipHostGetDevicePointer((void**)&d_osTail,(void*)osTail,0));
	checkCudaErrors(hipHostAlloc((void**)&osHead,sizeof(int),hipHostMallocMapped));
	checkCudaErrors(hipHostGetDevicePointer((void**)&d_osHead,(void*)osHead,0));

	//hipMalloc((void **)&osTail, sizeof(int)*1);
	//hipMalloc((void **)&osHead, sizeof(int)*1);
	hipMalloc((void **)&csTail, sizeof(int));
	hipMalloc((void **)&csHead, sizeof(int));
	hipMalloc((void **)&openSet, sizeof(CONFIGSTACK)*SETSIZE);
	hipMalloc((void **)&closeSet, sizeof(CONFIGSTACK)*SETSIZE);
	hipMemcpy(openSet, &initstate, sizeof(CONFIGSTACK), hipMemcpyHostToDevice);
	*osTail=1;
	*osHead=0;
	
	//declare the solution set
	int *solutionSize, solutionResultSize=0;
	CONFIGSTACK *solutionSet, *solutionResult;
	solutionResult=(CONFIGSTACK *)malloc(sizeof(CONFIGSTACK)*100);
	hipMalloc((void **)&solutionSize, sizeof(int));
	hipMalloc((void **)&solutionSet, sizeof(CONFIGSTACK)*100);

	//prepare the dag
	DAG_CUDA dag_d;
	hipMalloc((void **)&(dag_d.vertex), sizeof(int)*v_size);
	hipMalloc((void **)&(dag_d.type), sizeof(int)*v_size);
	hipMalloc((void **)&(dag_d.edge), sizeof(int)*2*ESIZE);

	float *priceOnDemand_d;
	hipMalloc((void **)&priceOnDemand_d, sizeof(float)*TYPES);
	hipMemcpy(priceOnDemand_d, priceOnDemand, sizeof(float)*TYPES, hipMemcpyHostToDevice);

	//prepare the *probestTime
	float *probestTime_a;
	checkCudaErrors(hipMalloc((void**)&probestTime_a, sizeof(float)*randomsize*TYPES*VSIZE));
	for (int i=0;i<VSIZE;++i)
	{
		//checkCudaErrors(
		hipMemcpy(&probestTime_a[i*randomsize*TYPES], probestTime[i], sizeof(float)*randomsize*TYPES,hipMemcpyHostToDevice);
	}

	hipMemcpy(dag_d.vertex, vertex, sizeof(int)*v_size, hipMemcpyHostToDevice);
	hipMemcpy(dag_d.edge, edge, 2*sizeof(int)*ESIZE, hipMemcpyHostToDevice);

	int *buffer_size, *d_buffer_size;

	checkCudaErrors(hipHostAlloc((void**)&buffer_size,sizeof(int),hipHostMallocMapped));
	checkCudaErrors(hipHostGetDevicePointer((void**)&d_buffer_size,(void*)buffer_size,0));

	CONFIGSTACK *buffer;//, *buffer_child;
	hipMalloc((void **)&buffer, sizeof(CONFIGSTACK)*buffer_size_max);
	//buffer_child=(CONFIGSTACK *)malloc(sizeof(CONFIGSTACK)*buffer_size_max*4);

	int zero=0;
	hipMemcpy(csHead, &zero, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(csTail, &zero, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(solutionSize, &zero, sizeof(int), hipMemcpyHostToDevice);

	int *gv, gv_h=(int)(globalBestCost*1000.0);
	hipMalloc((void **)&gv, sizeof(int));
	hipMemcpy(gv, &gv_h, sizeof(int), hipMemcpyHostToDevice);
	
	int searchcount=0;
	*buffer_size=1;
	
	int *d_inEdgeCount, *d_in_edge_own;
	int inEdgeCount[VSIZE], in_edge_own[VSIZE*ESIZE];
        for (int i=0;i<VSIZE;++i)
        {
            inEdgeCount[i]=0;
            for (int j=1;j<2*ESIZE;j+=2)
            {
                    if (edge[j]==i)
                    {
                            in_edge_own[inEdgeCount[i]+ESIZE*i]=edge[j-1];
                            inEdgeCount[i]++;
                    }
            }
        }
	hipMalloc((void**)&d_inEdgeCount, sizeof(int)*VSIZE);
	hipMalloc((void**)&d_in_edge_own, sizeof(int)*VSIZE*ESIZE);
	hipMemcpy(d_inEdgeCount, inEdgeCount, sizeof(int)*VSIZE, hipMemcpyHostToDevice);
	hipMemcpy(d_in_edge_own, in_edge_own, sizeof(int)*VSIZE*ESIZE, hipMemcpyHostToDevice);

	do
	{
		int sizeOpenSet=*osTail>=*osHead?(*osTail-*osHead):(SETSIZE-*osHead+*osTail);
		int length=(int)sizeOpenSet/buffer_size_max;
		length=length>1?length:1;
		length=length<32?length:32;
		top_k<<< buffer_size_max, length, 2*length>>>(d_buffer_size, buffer, openSet, d_osHead, d_osTail);
		hipDeviceSynchronize();
		
		evaluate<<<*buffer_size, 128>>>(buffer, dag_d, openSet, d_osHead, d_osTail, closeSet, csHead, csTail, solutionSet, solutionSize, 
				flag, types, probestTime_a, OnDemandLag, deadline, meet_dl, randomsize, priceOnDemand_d, globalBestCost, gv, d_inEdgeCount, d_in_edge_own);
		hipDeviceSynchronize();
		
		
		searchcount+=*buffer_size;
	}while(!(*osHead == *osTail || searchcount>10000));
		//update<<<>>>(buffer_child, );
	
	hipMemcpy(solutionResult, solutionSet, sizeof(CONFIGSTACK)*100, hipMemcpyDeviceToHost);
	hipMemcpy(&solutionResultSize, solutionSize, sizeof(int), hipMemcpyDeviceToHost);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipMemcpy(&gv_h, gv, sizeof(int), hipMemcpyDeviceToHost);
	printf ("CUDA function stoped after %d rounds with global best %f\n", searchcount, gv_h/1000.0);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf ("Elapsed Time %f ms. Solutions found: %d\n", elapsedTime, solutionResultSize);
	printf ("OpenSet: %d - %d\n",*osHead, *osTail );
	for (int i=0;i<solutionResultSize;++i)
	{
		for (int j=0;j<VSIZE;++j)
			printf ("%d ", solutionResult[i].configurations[j]);
		printf ("%f\n", solutionResult[i].fvalue);
	}

}

